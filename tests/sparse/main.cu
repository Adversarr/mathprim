#include <mathprim/sparse/basic_sparse.hpp>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

// CUDA 错误检查宏
#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
}

// cuSPARSE 错误检查宏
#define CHECK_CUSPARSE(call) { \
    hipsparseStatus_t status = call; \
    if (status != HIPSPARSE_STATUS_SUCCESS) { \
        printf("cuSPARSE Error at %s:%d - %d\n", __FILE__, __LINE__, status); \
        exit(1); \
    } \
}

int main() {
    // 初始化 cuSPARSE 句柄
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // 定义 CSR 矩阵示例 (3x3)
    const int rows = 3, cols = 3, nnz = 5;
    float h_csr_values[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};
    int h_csr_col_idx[] = {0, 1, 1, 2, 0};   // 列索引
    int h_csr_row_ptr[] = {0, 2, 4, 5};      // 行指针

    // 定义输入向量和输出向量 (全1向量)
    float h_x[] = {1.0f, 1.0f, 1.0f};
    float h_y[rows] = {0.0f};

    // 分配设备内存
    float *d_csr_values, *d_x, *d_y;
    int *d_csr_col_idx, *d_csr_row_ptr;

    CHECK_CUDA(hipMalloc(&d_csr_values, nnz * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_csr_col_idx, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_csr_row_ptr, (rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_x, cols * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_y, rows * sizeof(float)));

    // 拷贝数据到设备
    CHECK_CUDA(hipMemcpy(d_csr_values, h_csr_values, nnz * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csr_col_idx, h_csr_col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csr_row_ptr, h_csr_row_ptr, (rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, h_x, cols * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(d_y, 0, rows * sizeof(float))); // 初始化输出为0

    // 创建矩阵描述符
    hipsparseSpMatDescr_t matA;
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matA,                    // 矩阵描述符
        rows, cols, nnz,          // 行数、列数、非零元数
        d_csr_row_ptr,            // 行指针 (设备内存)
        d_csr_col_idx,            // 列索引 (设备内存)
        d_csr_values,             // 值数组 (设备内存)
        HIPSPARSE_INDEX_32I,       // 行指针索引类型 (32位整数)
        HIPSPARSE_INDEX_32I,       // 列索引类型 (32位整数)
        HIPSPARSE_INDEX_BASE_ZERO, // 索引基址 (从0开始)
        HIP_R_32F                // 数据类型 (32位浮点)
    ));

    // 创建向量描述符 (输入x和输出y)
    hipsparseDnVecDescr_t vecX, vecY;
    CHECK_CUSPARSE(hipsparseCreateDnVec(
        &vecX,                    // 向量描述符
        cols,                     // 向量长度
        d_x,                      // 数据指针 (设备内存)
        HIP_R_32F                // 数据类型
    ));
    CHECK_CUSPARSE(hipsparseCreateDnVec(
        &vecY,
        rows,
        d_y,
        HIP_R_32F
    ));

    // 执行 SpMV: y = alpha * A * x + beta * y
    float alpha = 1.0f, beta = 0.0f;
    size_t bufferSize = 0;
    void* d_buffer = nullptr;

    // 获取所需缓冲区大小
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, // 矩阵不转置
        &alpha,
        matA,
        vecX,
        &beta,
        vecY,
        HIP_R_32F,                       // 计算数据类型
        HIPSPARSE_SPMV_ALG_DEFAULT,        // 算法选择
        &bufferSize
    ));

    // 分配临时缓冲区
    CHECK_CUDA(hipMalloc(&d_buffer, bufferSize));

    // 执行 SpMV 计算
    CHECK_CUSPARSE(hipsparseSpMV(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha,
        matA,
        vecX,
        &beta,
        vecY,
        HIP_R_32F,
        HIPSPARSE_SPMV_ALG_DEFAULT,
        d_buffer
    ));

    // 拷贝结果回主机
    CHECK_CUDA(hipMemcpy(h_y, d_y, rows * sizeof(float), hipMemcpyDeviceToHost));

    // 打印结果
    printf("Result y = A*x:\n");
    for (int i = 0; i < rows; ++i) {
        printf("y[%d] = %.2f\n", i, h_y[i]);
    }
    // 预期输出:
    // y[0] = 1*1 + 2*1 = 3.00
    // y[1] = 3*1 + 4*1 = 7.00
    // y[2] = 5*1 = 5.00

    // 释放资源
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    CHECK_CUDA(hipFree(d_buffer));
    CHECK_CUDA(hipFree(d_csr_values));
    CHECK_CUDA(hipFree(d_csr_col_idx));
    CHECK_CUDA(hipFree(d_csr_row_ptr));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));

    return 0;
}