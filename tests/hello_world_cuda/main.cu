#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cstdio>
#include <iostream>

#define MATHPRIM_VERBOSE_MALLOC 1
#include <mathprim/core/buffer.hpp>
#include <mathprim/core/devices/cuda.cuh>
#include <mathprim/parallel/cuda.cuh>
#include <mathprim/supports/stringify.hpp>

using namespace mathprim;
using namespace mathprim::literal;

__global__ void set_value(float *ptr, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    ptr[idx] = static_cast<float>(idx);
    printf("ptr[%d] = %f\n", idx, static_cast<float>(idx));
  }
}

__global__ void get_value(field_t<cuda_vec4f32_const_view_t> view) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx == 0) {
    printf("view.size() = %d\n", view.size());
    printf("view.shape() = (%d, %d)\n", view.shape(0), view.shape(1));
  }

  if (idx < view.size()) {
    auto i = idx / 4;
    auto j = idx % 4;
    printf("view[%d][%d] = %f\n", i, j, view(i, j));
  }
}

int main() {
  auto buf = make_buffer<float, device::cuda>(10, 4_s);
  auto view = buf.view();
  std::cout << view.size() << std::endl;
  auto [i, j] = view.shape();
  std::cout << i << " " << j << std::endl;
  set_value<<<view.size(), 1>>>(buf.data(), buf.size());
  get_value<<<view.size(), 1>>>(view);

  par::cuda parfor;

  parfor.run(view.shape(), [view] __device__(index_array<2> idx) {
    auto [i, j] = idx;
    printf("Lambda view[%d, %d] = %f\n", i, j, view(i, j));
  });

  parfor.run(dshape<4>(10, 4, 1, 1), [view] __device__(index_array<4> idx) {
    auto [i, j, k, l] = idx;
    printf("Lambda view[%d, %d, %d, %d] = %f\n", i, j, k, l, view(i, j));
  });

  // Allocate a pitch memory
  float *ptr = nullptr;
  size_t pitch = 0;
  // hipMallocPitch use [weight, height] as parameter
  size_t width = 4 * sizeof(float), height = 10 * sizeof(float);
  hipMallocPitch(&ptr, &pitch, width, height);
  auto pitched_ptr_cuda = make_hipPitchedPtr(ptr, pitch, width, height);
  std::cout << "pitched_ptr_cuda.ptr = " << pitched_ptr_cuda.ptr
            << ", pitched_ptr_cuda.pitch = " << pitched_ptr_cuda.pitch
            << ", pitched_ptr_cuda.xsize = " << pitched_ptr_cuda.xsize
            << ", pitched_ptr_cuda.ysize = " << pitched_ptr_cuda.ysize
            << std::endl;
  // create view.
  auto view_pitched = from_cuda_pitched_ptr<float>(pitched_ptr_cuda);
  std::cout << "view_pitched=" << view_pitched << std::endl;

  // view back.
  auto pitched_ptr_cuda_back = to_cuda_pitched_ptr(view_pitched);
  std::cout << "pitched_ptr_cuda_back.ptr = " << pitched_ptr_cuda_back.ptr
            << ", pitched_ptr_cuda_back.pitch = " << pitched_ptr_cuda_back.pitch
            << ", pitched_ptr_cuda_back.xsize = " << pitched_ptr_cuda_back.xsize
            << ", pitched_ptr_cuda_back.ysize = " << pitched_ptr_cuda_back.ysize
            << std::endl;

  // Free the memory
  hipFree(ptr);

  // Make a pitched buffer
  auto pitched_buf = make_cuda_pitched_buffer<float>(make_shape(10, 4));
  std::cout << "pitched_buf=" << pitched_buf.view() << std::endl;

  hipDeviceSynchronize();

  // cuda streams.
  hipStream_t stream; hipStreamCreate(&stream);

  par::cuda parfor_stream(stream);
  par::cuda parfor_default;
  parfor_stream.run(view.shape(), [view] __device__(index_array<2> idx) {
    auto [i, j] = idx;
    printf("Lambda streamd view[%d, %d] = %f\n", i, j, view(i, j));
  });
  parfor_default.run(view.shape(), [view] __device__(index_array<2> idx) {
    auto [i, j] = idx;
    printf("Lambda default view[%d, %d] = %f\n", i, j, view(i, j));
  });

  parfor_stream.sync();
  parfor_default.sync();
  hipStreamDestroy(stream);

  // vmap:
  auto buf2 = make_buffer<float, device::cuda>(10);

  parfor_default.vmap(
      [] __device__(auto vec4, auto &out) {
        out = vec4[0] + vec4[1] + vec4[2] + vec4[3];
      },
      buf.view(), buf2.view());

  parfor_default.run(buf2.shape(), [view = buf2.view()] __device__(index_t i) {
    // Should be 6, 22, 38, 54, 70, 86, 102, 118, 134, 150 = 16 i + 6
    if (view[i] != 16 * i + 6) {
      printf("Error: buf2[%d] = %f\n", i, view(i));
    } else {
      printf("Ok: buf2[%d] = %f\n", i, view(i));
    }
  });
  parfor_default.sync();

  return EXIT_SUCCESS;
}
